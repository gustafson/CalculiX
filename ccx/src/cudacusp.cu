
#include <hip/hip_runtime.h>
/*     CalculiX - A 3-dimensional finite element program                 */
/*              Copyright (C) 1998-2011 Guido Dhondt                     */
/*     This subroutine                                                   */
/*              Copyright (C) 2013 Peter A. Gustafson                    */
/*                                                                       */
/*     This program is free software; you can redistribute it and/or     */
/*     modify it under the terms of the GNU General Public License as    */
/*     published by the Free Software Foundation(version 2);    */
/*                                                                       */

/*     This program is distributed in the hope that it will be useful,   */
/*     but WITHOUT ANY WARRANTY; without even the implied warranty of    */ 
/*     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the      */
/*     GNU General Public License for more details.                      */

/*     You should have received a copy of the GNU General Public License */
/*     along with this program; if not, write to the Free Software       */
/*     Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.         */

#ifdef CUDACUSP

#include <cusp/hyb_matrix.h>
#include <cusp/dia_matrix.h>
// #include <cusp/gallery/poisson.h>
#include <cusp/krylov/cg.h>
// #include <cusp/krylov/cg_m.h>
// #include <cusp/krylov/bicg.h>
// #include <cusp/krylov/bicgstab.h>
#include <cusp/version.h>
// #include <cusp/print.h>
#include <cusp/array1d.h>
#include <cusp/multiply.h>
#include <cusp/precond/ainv.h> 
#include <iostream>
#include <cusp/precond/smoothed_aggregation.h>
// #include <cusp/krylov/gmres.h>
// #include <cusp/detail/format_utils.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
// #include <cusp/ell_matrix.h>


// which floating point type to use
typedef double ValueType;
// typedef cusp::host_memory MemorySpace;
typedef cusp::device_memory MemorySpace;

template <typename T>
struct invsqr : public thrust::unary_function<T,T>
{
  __host__ __device__
  T operator()(const T& v) 
  {
    return T (1.0)/sqrt(v);
  }
};

template <typename T>
struct absolute : public thrust::unary_function<T,T>
{
    __host__ __device__
    T operator()(T x)
  {
    return x < 0 ? -x : x;
  }
};

extern "C"
int cudacusp(double *ad, double *au, double *adb, double *aub, double *sigma, 
	     double *b, int *icol, int *irow, int *neq, int *nzs, 
	     int *symmetryflag, int *inputformat, int *jq, int *nzs3)
{
  int cuda_major =  CUDA_VERSION / 1000;
  int cuda_minor = (CUDA_VERSION % 1000) / 10;

  int thrust_major = THRUST_MAJOR_VERSION;
  int thrust_minor = THRUST_MINOR_VERSION;

  int cusp_major = CUSP_MAJOR_VERSION;
  int cusp_minor = CUSP_MINOR_VERSION;

  clock_t timeb;
  clock_t timee;

  std::cout << " Using CUDA based on CUSP CG SOLVER\n";
  std::cout << "   CUDA   v" << cuda_major   << "." << cuda_minor   << "\n";
  std::cout << "   Thrust v" << thrust_major << "." << thrust_minor << "\n";
  std::cout << "   Cusp   v" << cusp_major   << "." << cusp_minor   << "\n";


  timeb = clock();
  // Test for non zero values
  int nvals=0;
  for (int i=0; i<*neq; i++){if (ad[i]<0) nvals++;}
  if (nvals) {thrust::transform(ad, ad+*neq, ad, absolute<ValueType>());}

  /* Fill the matrix.  
     The off diagonal triangle is columnar from ccx
     irow() identifies the row within the column
     icol() identifies the number of non zeros within the column
     Move the the next column after achieving icol() within a column. */
  cusp::coo_matrix<int, ValueType, cusp::host_memory> A(*neq,*neq,2*(*nzs)+*neq);
  // ASSEMBLE FULL MATRIX.  No symmetric matrix defined in CUSP //
  { // Scope for matrix assembly
    int k=0; // data index
    int l=0; // row index
    int m=0; // column tracker index
  
    // This is somewhat expensive... can it be parallelized.
    for (int i = 0; i < *neq; i++){
      // This is for the diagonal
      A.row_indices[k] = i; 
      A.column_indices[k] = i; 
      A.values[k++] = ad[i];

      for (int j = 0; j < icol[i]; j++){
      // This is for the off-diagonals
	int n = irow[m]-1;
	A.row_indices[k] = l; 
	A.column_indices[k] = n; 
	A.values[k++] = au[m];
	A.row_indices[k] = n; 
	A.column_indices[k] = l; 
	A.values[k++] = au[m++];
      }
      l++;
    }
  }

  A.sort_by_row_and_column();
  // cusp::print(A);
  cusp::hyb_matrix<int, ValueType, MemorySpace> AA = A;
  timee = clock();
  std::cout << "  Assembled stiffness matrix on CUDA device in = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << "\n\n";

  timee = clock();
  
  timeb = clock();
  printf ("Diagnonal preconditioner\n");
  cusp::precond::diagonal<ValueType, MemorySpace> MM(AA);
  timee = clock();
  std::cout << "  Preconditioning time = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << "\n\n";
  
  // allocate storage for and copy right hand side (BB). 
  cusp::array1d<ValueType, MemorySpace> BB(*neq, 0.0);
  thrust::copy (b, b+*neq, BB.begin());
  
  timeb = clock();
  // set stopping criteria 
  // http://docs.cusp-library.googlecode.com/hg/classcusp_1_1default__monitor.html
  // ||b - A x|| <= absolute_tolerance + relative_tolerance * ||b||
  
  int i=50000;
  // if ((*b)<0.0){
  if (nvals){
    // Non-positive definite.  Give up quickly after spawning an answer
    // thrust::copy (ad, ad+*neq, DD.begin());
    // thrust::transform(DD.begin(), DD.end(), DD.begin(), absolute<ValueType>());
    i=0;
    printf ("There are %i negative values on the diagonal.  The attempt is abandoned.\n", nvals);
  }
  cusp::verbose_monitor<ValueType> monitor(BB, i, 1e-6);


  // solve the linear system AA * XX = BB 
  cusp::krylov::cg(AA, BB, BB, monitor, MM); //Conjugate Gradient method
  timee = clock();

  std::cout << "  CUDA iterative solver time = " << 
    (double(timee)-double(timeb))/double(CLOCKS_PER_SEC) << "\n\n";

  // Copy the result to the b array
  thrust::copy (BB.begin(), BB.end(), b);

  if (!monitor.converged()){
    printf (" WARNING: Cuda Cusp did not find a solution.\n");
  }
  return 0;
}
#endif

